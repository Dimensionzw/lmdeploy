#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.
#include "src/turbomind/models/llama/rotary_emb.h"
#include <map>

namespace turbomind {

__device__ int get_batch_id(int qi, int* q_len, int batch_size)
{
    int result{};
    int end = (batch_size + blockDim.x - 1) / blockDim.x * blockDim.x;
    for (int i = threadIdx.x; i < end; i += blockDim.x) {
        int  prefix_sum = (i < batch_size) ? q_len[i + 1] : q_len[batch_size];
        auto count      = __syncthreads_count(prefix_sum > qi);
        if (count != 0) {
            result = i / blockDim.x * blockDim.x + blockDim.x - count + 1;
            break;
        }
    }
    return result;
}

__inline__ __device__ float compute_default_parameters(float base, float dim, int di, float factor)
{
    float scale_factor = -log2f(base) / dim;
    float inv_freq     = exp2f(di * scale_factor) * factor;
    return inv_freq;
}

__global__ void computeCosSinDefault(const float* rope_base,
                                     int*         q_len,
                                     int*         k_len,
                                     int          token_num,
                                     int          batch_size,
                                     int          dim,
                                     float        factor,
                                     float*       cos_sin)
{
    int qi = blockIdx.x;
    int di = threadIdx.x;

    int   bid         = get_batch_id(qi, q_len, batch_size);
    int   history_len = (k_len[bid] - k_len[bid - 1]) - (q_len[bid] - q_len[bid - 1]);
    float base        = rope_base[bid - 1];
    float ti          = history_len + qi - q_len[bid - 1];

    float inv_freq = compute_default_parameters(base, dim, di * 2, factor);
    float c, s;
    sincosf(ti * inv_freq, &s, &c);
    (float2&)cos_sin[dim * qi + 2 * di] = {c, s};
}

__global__ void computeCosSinLlama3(const float* rope_base,
                                    int*         q_len,
                                    int*         k_len,
                                    int          token_num,
                                    int          batch_size,
                                    int          dim,
                                    float        llama3_inv_scaling_factor,
                                    float        llama3_alpha,
                                    float        llama3_beta,
                                    float*       cos_sin)
{
    int qi = blockIdx.x;
    int di = threadIdx.x;

    int   bid         = get_batch_id(qi, q_len, batch_size);
    int   history_len = (k_len[bid] - k_len[bid - 1]) - (q_len[bid] - q_len[bid - 1]);
    float base        = rope_base[bid - 1];
    float ti          = history_len + qi - q_len[bid - 1];

    float inv_freq = compute_default_parameters(base, dim, di * 2, 1.0f);
    auto  smooth   = fmaxf(0.f, fminf(1.f, llama3_alpha * inv_freq - llama3_beta));
    inv_freq       = (1 - smooth) * inv_freq * llama3_inv_scaling_factor + smooth * inv_freq;
    float c, s;
    sincosf(ti * inv_freq, &s, &c);
    (float2&)cos_sin[dim * qi + 2 * di] = {c, s};
}

__global__ void computeCosSinYarn(const float* rope_base,
                                  int*         q_len,
                                  int*         k_len,
                                  int          token_num,
                                  int          batch_size,
                                  int          dim,
                                  float        yarn_ramp_inv_factor_div_2,
                                  float        yarn_ramp_inv_factor_mul_min,
                                  float        yarn_inv_scaling_factor,
                                  float        attention_scaling,
                                  float*       cos_sin)
{
    int qi = blockIdx.x;
    int di = threadIdx.x;

    int   bid         = get_batch_id(qi, q_len, batch_size);
    int   history_len = (k_len[bid] - k_len[bid - 1]) - (q_len[bid] - q_len[bid - 1]);
    float base        = rope_base[bid - 1];
    float ti          = history_len + qi - q_len[bid - 1];

    float inv_freq = compute_default_parameters(base, dim, di * 2, 1.0f);
    float alpha    = 2 * di * yarn_ramp_inv_factor_div_2 - yarn_ramp_inv_factor_mul_min;
    alpha          = fmaxf(0.f, fminf(1.f, alpha));
    inv_freq       = inv_freq - inv_freq * alpha * yarn_inv_scaling_factor;

    float c, s;
    sincosf(ti * inv_freq, &s, &c);
    c *= attention_scaling;
    s *= attention_scaling;
    (float2&)cos_sin[dim * qi + 2 * di] = {c, s};
}

RotaryScalingType GetRoPEType(const std::string& type)
{
    std::map<std::string, RotaryScalingType> lookup = {{"", RotaryScalingType::kDefault},
                                                       {"linear", RotaryScalingType::kLinear},
                                                       {"dynamic", RotaryScalingType::kDynamic},
                                                       {"yarn", RotaryScalingType::kYarn},
                                                       {"llama3", RotaryScalingType::kLlama3}};
    return lookup.at(type);
}

void RotaryEmbeddingV2::freeBuffer()
{
    allocator_->free((void**)&cos_sin_);
}

void RotaryEmbeddingV2::allocateBuffer(size_t token_num)
{
    cos_sin_ = (float*)allocator_->reMalloc(cos_sin_, sizeof(float) * token_num * dim_);
}

RotaryEmbeddingV2::RotaryEmbeddingV2(const AttentionParam& param, hipStream_t stream, IAllocator* allocator):
    stream_(stream), allocator_(allocator)
{
    type_ = param.rope.type;
    dim_  = param.rope.dim;

    switch (type_) {
        case RotaryScalingType::kDefault:
            break;
        case RotaryScalingType::kLinear:
            inv_factor_ = 1.0f / param.rope.factor;
            break;
        case RotaryScalingType::kDynamic:
            inv_factor_ = param.rope.factor;
            break;
        case RotaryScalingType::kYarn: {
            const double PI                  = 3.14159265358979323846;
            auto         find_correction_dim = [&](float num_rotations) {
                return (param.rope.dim * std::log(param.rope.max_position_embeddings / (num_rotations * 2 * PI)))
                       / (2 * std::log(param.rope.base));
            };
            auto find_correction_range = [&](float low_rot, float high_rot, float& low, float& high) {
                low  = std::floor(find_correction_dim(low_rot));
                high = std::ceil(find_correction_dim(high_rot));
                low  = std::max(low, 0.f);
                high = std::min(high, param.rope.dim - 1.f);
            };
            float low, high;
            find_correction_range(param.rope.yarn.beta_fast, param.rope.yarn.beta_slow, low, high);
            if (low == high) {
                high += 0.01f;
            }
            yarn_.yarn_ramp_inv_factor_div_2   = 1.0 / (high - low) / 2.0;
            yarn_.yarn_ramp_inv_factor_mul_min = 1.0 / (high - low) * low;
            yarn_.yarn_inv_scaling_factor      = (1 - 1.0 / param.rope.factor);
            yarn_.attention_factor             = param.rope.yarn.attention_factor;
            break;
        }
        case RotaryScalingType::kLlama3: {
            const double PI            = 3.14159265358979323846;
            float inv_diff_freq_factor = 1.0 / (param.rope.llama3.high_freq_factor - param.rope.llama3.low_freq_factor);
            llama3_.llama3_inv_scaling_factor = 1.0 / param.rope.factor;
            llama3_.llama3_alpha = param.rope.llama3.original_max_position_embeddings / (2 * PI) * inv_diff_freq_factor;
            llama3_.llama3_beta  = param.rope.llama3.low_freq_factor * inv_diff_freq_factor;
            break;
        }
        default:
            FT_CHECK(0);
            break;
    }
}

void RotaryEmbeddingV2::forward(const RotaryEmbeddingV2Params& params)
{
    allocateBuffer(params.token_num);

    const int grid  = params.token_num;
    const int block = dim_ / 2;

    switch (type_) {
        case RotaryScalingType::kDefault:
        case RotaryScalingType::kLinear:
        case RotaryScalingType::kDynamic:
            computeCosSinDefault<<<grid, block, 0, stream_>>>(params.rope_theta,
                                                              params.q_len,
                                                              params.k_ken,
                                                              params.token_num,
                                                              params.batch_size,
                                                              dim_,
                                                              inv_factor_,
                                                              cos_sin_);
            break;
        case RotaryScalingType::kLlama3:
            computeCosSinLlama3<<<grid, block, 0, stream_>>>(params.rope_theta,
                                                             params.q_len,
                                                             params.k_ken,
                                                             params.token_num,
                                                             params.batch_size,
                                                             dim_,
                                                             llama3_.llama3_inv_scaling_factor,
                                                             llama3_.llama3_alpha,
                                                             llama3_.llama3_beta,
                                                             cos_sin_);
            break;
        case RotaryScalingType::kYarn:
            computeCosSinYarn<<<grid, block, 0, stream_>>>(params.rope_theta,
                                                           params.q_len,
                                                           params.k_ken,
                                                           params.token_num,
                                                           params.batch_size,
                                                           dim_,
                                                           yarn_.yarn_ramp_inv_factor_div_2,
                                                           yarn_.yarn_ramp_inv_factor_mul_min,
                                                           yarn_.yarn_inv_scaling_factor,
                                                           yarn_.attention_factor,
                                                           cos_sin_);
            break;
        default:
            FT_CHECK(0);
    }
}

}  // namespace turbomind
